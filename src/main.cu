#include <chrono>
#include <cstring>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#define DATA_SIZE 1024 * 1024 * 256
#define SRC_GPU 0
#define DEST_GPU 1

#define CHECK_CUDA(call)                                                       \
  {                                                                            \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(1);                                                                 \
    }                                                                          \
  }

long openmp_sum(int *data, size_t size) {
  long sum = 0;
  int num_threads = omp_get_max_threads();
#pragma omp parallel for simd reduction(+ : sum) num_threads(num_threads)
  for (size_t i = 0; i < size; i++) {
    sum += data[i];
  }

  return sum;
}

// Transfer data host-to-device or device-to-host async
void transfer_data(int gpu_id, int *src_data, int *host_buffer,
                   size_t data_size, hipStream_t stream,
                   hipEvent_t **timing_events, bool dtoh = true) {
  CHECK_CUDA(hipSetDevice(gpu_id));

  hipEvent_t *events = (hipEvent_t *)malloc(2 * sizeof(hipEvent_t));

  CHECK_CUDA(hipEventCreate(&events[0]));
  CHECK_CUDA(hipEventCreate(&events[1]));

  CHECK_CUDA(hipEventRecord(events[0], stream));
  if (dtoh) {
    CHECK_CUDA(hipMemcpyAsync(host_buffer, src_data, data_size,
                               hipMemcpyDeviceToHost, stream));
  } else {
    CHECK_CUDA(hipMemcpyAsync(src_data, host_buffer, data_size,
                               hipMemcpyHostToDevice, stream));
  }
  CHECK_CUDA(hipEventRecord(events[1], stream));
  *timing_events = events;
}

// TODO: Validate data on destination

// Generate random data on CPU and send it to GPU with id i
void generate_data(int gpu_id, int *host_buffer, int *gpu_buffer,
                   size_t data_size, hipStream_t stream) {
  // Generate random data on CPU
  //
  hipEvent_t *timing_events;
  for (int j = 0; j < data_size / sizeof(int); j++) {
    host_buffer[j] = rand();
  }
  // Transfer data to GPU
  transfer_data(gpu_id, gpu_buffer, host_buffer, data_size, stream,
                &timing_events, false);
}

int main(int argc, char **argv) {

  CHECK_CUDA(hipSetDevice(SRC_GPU));
  int *data = (int *)malloc(DATA_SIZE * sizeof(int));
  int *gpu_data;
  CHECK_CUDA(hipMalloc((void **)&gpu_data, DATA_SIZE * sizeof(int)));

  hipStream_t src_stream, dest_stream;

  CHECK_CUDA(hipStreamCreate(&src_stream));

  generate_data(0, data, gpu_data, DATA_SIZE * sizeof(int), src_stream);

  memset(data, 0, DATA_SIZE * sizeof(int));

  hipEvent_t *timing_events_src_host;
  printf("Data generated on src GPU, sending from src to host\n");
  transfer_data(SRC_GPU, gpu_data, data, DATA_SIZE * sizeof(int), src_stream,
                &timing_events_src_host);


  auto start = std::chrono::high_resolution_clock::now();
  long result = openmp_sum(data, DATA_SIZE);
  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

  printf("Timing for OpenMP sum: %ld\n", duration.count());

  CHECK_CUDA(hipSetDevice(DEST_GPU));
  CHECK_CUDA(hipStreamCreate(&dest_stream));

  hipEvent_t *timing_events_host_dest;
  printf("Data received on host, sending from host to dest\n");
  transfer_data(DEST_GPU, gpu_data, &result, sizeof(long), dest_stream,
                &timing_events_host_dest, false);

  CHECK_CUDA(hipSetDevice(SRC_GPU));
  CHECK_CUDA(hipStreamSynchronize(src_stream));
  CHECK_CUDA(hipSetDevice(DEST_GPU));
  CHECK_CUDA(hipStreamSynchronize(dest_stream));

  float src_host_timing, host_dest_timing;

  CHECK_CUDA(hipEventElapsedTime(&src_host_timing, timing_events_src_host[0],
                                  timing_events_src_host[1]));
  CHECK_CUDA(hipEventElapsedTime(&host_dest_timing, timing_events_host_dest[0],
                                  timing_events_host_dest[1]));

  printf("Src to Host: %f\n", src_host_timing);
  printf("Host to Dest: %f\n", host_dest_timing);

  return 0;
}
