#include "hip/hip_runtime.h"
#include "distinct_merge.h"
#include "utils.h"
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <math.h>
#include <mutex>
#include <thread>

DistinctMerge::DistinctMerge(const std::vector<int *> receive_buffers,
                             const std::vector<int> receive_buffer_sizes)
    : receive_buffers(receive_buffers),
      receive_buffer_sizes(receive_buffer_sizes) {

  // print the receive buffers
  // std::cout << "Receive buffers: " << std::endl;
  // for (int i = 0; i < receive_buffers.size(); i++) {
  //   std::cout << "Buffer " << i << ": ";
  //   for (int j = 0; j < receive_buffer_sizes[i]; j++) {
  //     std::cout << receive_buffers[i][j] << " ";
  //   }
  //   std::cout << std::endl;
  // }

  //std::thread sender_thread(&DistinctMerge::sender, this);
}

int DistinctMerge::check_value(int value) {
  // WARN: We should remove locking later since its a performance bottleneck (we
  // should use somthing like Intel TBB)

  std::unique_lock<std::mutex> lock(this->seen_values_mutex);

  auto it = seen_values.find(value);
  if (it != seen_values.end()) {
    // INFO: We assume that input data are positive integers
    lock.unlock();
    return -1;
  } else {
    seen_values[value] = true;
    lock.unlock();
    return value;
  }
}

bool DistinctMerge::stage(int value) {

  std::unique_lock<std::mutex> lock(this->send_buffer_mutex);

  this->send_buffer[this->send_buffer_end_index++] = value;

  lock.unlock();

  return true;
}

void DistinctMerge::sender() {
  // TODO: this function check the send buffer and sends data whenever it
  // reached the threshold

  while (true) {
    int difference =
        std::abs(this->send_buffer_start_index - this->send_buffer_end_index);
    if (difference >= DISTINCT_MERGE_BUFFER_THRESHOLD) {
      std::cout << "Sending data" << std::endl;
      this->send_buffer_start_index += difference;
    }
  }
}

DistinctMergeGPU::DistinctMergeGPU(int gpu_id, int tuples_count, int chunk_size)
    : gpu_id(gpu_id), tuples_count(tuples_count), chunk_size(chunk_size) {
  // TODO: init random data on gpu
  CHECK_CUDA(hipSetDevice(gpu_id));
  CHECK_CUDA(hipMalloc((void **)&this->gpu_data, tuples_count * sizeof(int)));
  generate_data(gpu_id, this->gpu_data, tuples_count);

  // TODO: allocate destination buffer on cpu
  this->destination_buffer = new int[tuples_count];
}

void DistinctMergeGPU::exec(int start_index) {
  // TODO: Run the deduplication on the chunk (do it later, for now we just
  // assume that all tuples have unique values)

  // TODO: Send the deduplicated chunk to CPU
  CHECK_CUDA(hipMemcpy(
      this->destination_buffer + start_index, this->gpu_data + start_index,
      this->chunk_size * sizeof(int), hipMemcpyDeviceToHost));

  // TODO: Check the values and stage them for sending
  for (int i = start_index; i < start_index + this->chunk_size; i++) {
    int checked_value =
        this->cpu_merger->check_value(this->destination_buffer[i]);

    // Tuple is new so we should stage it into the send buffer
    if (checked_value != -1)
      this->cpu_merger->stage(checked_value);
  }
}

void DistinctMergeGPU::start() {
  std::cout << std::unitbuf;
  std::cout << "Starting GPU merger" << std::endl;

  int threads_count = this->tuples_count / this->chunk_size;

  std::thread threads[threads_count];

  for (int i = 0; i < threads_count; i++) {
    threads[i] =
        std::thread(&DistinctMergeGPU::exec, this, i * this->chunk_size);
  }

  for (int i = 0; i < threads_count; i++) {
    threads[i].join();
  }
}
